#include "ODE_LinAlg.cuh"

__host__ void SetNewtonTolerance()
{
    AbsTol_h = (1e-3) * RelTol;
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(AbsTol), &AbsTol_h, sizeof(double)));
    hipDeviceSynchronize();
}

