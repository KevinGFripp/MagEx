#include "Device_State_Functions.cuh"
#include <>
#include "Device_Globals_Constants.cuh"
#include "Host_Globals.cuh"
#include <hip/hip_runtime_api.h>
__host__ void UpdateDeviceTime(double time)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(t_d), &time, sizeof(double)));
}
__host__ void SetCurrentTime(double time)
{
    t_h = time;
    return;
}