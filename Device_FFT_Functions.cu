#include "Device_FFT_Functions.cuh"
#include <hipfft/hipfft.h>
#include "Device_Globals_Constants.cuh"
#include "Host_Globals.cuh"
#include "GlobalDefines.cuh"
#include "Pointer_Functions.cuh"
#include <hip/hip_runtime_api.h>
#include <>
#include "DemagnetisingTensor_Functions.cuh"

__host__ void FFTPlansInitialise(PLANS P)
{
    int Nx = 2 * NUM_h;
    int Ny = 2 * NUMY_h;
    int Nz = 2 * NUMZ_h;

    if (IsPBCEnabled == false)
    {
        Nx = 2 * NUM_h;
        Ny = 2 * NUMY_h;
        Nz = 2 * NUMZ_h;
    }
    if (PBC_x_h != 0)
    {
        Nx = NUM_h;
        Ny = 2 * NUMY_h;
        Nz = 2 * NUMZ_h;
    }
    if (PBC_y_h != 0)
    {
        Nx = 2 * NUM_h;
        Ny = NUMY_h;
        Nz = 2 * NUMZ_h;
    }
    if (PBC_y_h != 0 && PBC_x_h != 0)
    {
        Nx = NUM_h;
        Ny = NUMY_h;
        Nz = 2 * NUMZ_h;
    }

    int FFT_SIZE = Nx * Ny * Nz;

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(FFT_NORM), &FFT_SIZE, sizeof(int)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(PADNUM), &Nx, sizeof(int)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(PADNUMY), &Ny, sizeof(int)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(PADNUMZ), &Nz, sizeof(int)));

    CHECK_CUFFT_ERRORS(
        hipfftPlan3d(&(P->MxPlan), Nx, Ny, Nz, HIPFFT_D2Z));
    CHECK_CUFFT_ERRORS(
        hipfftPlan3d(&(P->MyPlan), Nx, Ny, Nz, HIPFFT_D2Z));
    CHECK_CUFFT_ERRORS(
        hipfftPlan3d(&(P->MzPlan), Nx, Ny, Nz, HIPFFT_D2Z));

    CHECK_CUFFT_ERRORS(
        hipfftPlan3d(&(P->HxPlan), Nx, Ny, Nz, HIPFFT_Z2D));
    CHECK_CUFFT_ERRORS(
        hipfftPlan3d(&(P->HyPlan), Nx, Ny, Nz, HIPFFT_Z2D));
    CHECK_CUFFT_ERRORS(
        hipfftPlan3d(&(P->HzPlan), Nx, Ny, Nz, HIPFFT_Z2D));

    PointerPlansCheck(P->MxPlan != HIPFFT_SUCCESS), PointerPlansCheck(P->MyPlan != HIPFFT_SUCCESS),
        PointerPlansCheck(P->MzPlan != HIPFFT_SUCCESS), PointerPlansCheck(P->HxPlan != HIPFFT_SUCCESS),
        PointerPlansCheck(P->HyPlan != HIPFFT_SUCCESS), PointerPlansCheck(P->HzPlan != HIPFFT_SUCCESS);

}
__host__ void FFTPlansInitialise_SinglePrecision_R2C(PLANS P)
{
    int Nx = 2 * NUM_h;
    int Ny = 2 * NUMY_h;
    int Nz = 2 * NUMZ_h;

    if (IsPBCEnabled == false)
    {
        Nx = 2 * NUM_h;
        Ny = 2 * NUMY_h;
        Nz = 2 * NUMZ_h;
    }
    if (PBC_x_h != 0)
    {
        Nx = NUM_h;
        Ny = 2 * NUMY_h;
        Nz = 2 * NUMZ_h;
    }
    if (PBC_y_h != 0)
    {
        Nx = 2 * NUM_h;
        Ny = NUMY_h;
        Nz = 2 * NUMZ_h;
    }
    if (PBC_y_h != 0 && PBC_x_h != 0)
    {
        Nx = NUM_h;
        Ny = NUMY_h;
        Nz = 2 * NUMZ_h;
    }

    int FFT_SIZE = Nx * Ny * Nz;

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(FFT_NORM), &FFT_SIZE, sizeof(int)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(PADNUM), &Nx, sizeof(int)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(PADNUMY), &Ny, sizeof(int)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(PADNUMZ), &Nz, sizeof(int)));

    CHECK_CUFFT_ERRORS(
        hipfftPlan3d(&(P->MxPlan), Nx, Ny, Nz, HIPFFT_R2C));
    CHECK_CUFFT_ERRORS(
        hipfftPlan3d(&(P->MyPlan), Nx, Ny, Nz, HIPFFT_R2C));
    CHECK_CUFFT_ERRORS(
        hipfftPlan3d(&(P->MzPlan), Nx, Ny, Nz, HIPFFT_R2C));

    CHECK_CUFFT_ERRORS(
        hipfftPlan3d(&(P->HxPlan), Nx, Ny, Nz, HIPFFT_C2R));
    CHECK_CUFFT_ERRORS(
        hipfftPlan3d(&(P->HyPlan), Nx, Ny, Nz, HIPFFT_C2R));
    CHECK_CUFFT_ERRORS(
        hipfftPlan3d(&(P->HzPlan), Nx, Ny, Nz, HIPFFT_C2R));

    PointerPlansCheck(P->MxPlan != HIPFFT_SUCCESS), PointerPlansCheck(P->MyPlan != HIPFFT_SUCCESS),
    PointerPlansCheck(P->MzPlan != HIPFFT_SUCCESS), PointerPlansCheck(P->HxPlan != HIPFFT_SUCCESS),
    PointerPlansCheck(P->HyPlan != HIPFFT_SUCCESS), PointerPlansCheck(P->HzPlan != HIPFFT_SUCCESS);

}
__host__ void DemagFieldInverseFFT_SinglePrecision_R2C(PLANS P)
{
    CHECK_CUFFT_ERRORS(
        hipfftExecC2R(P->HxPlan, ((DEVICE_PTR_STRUCT.DATA)->xFFT_s),
            ((DEVICE_PTR_STRUCT.DATA)->Outx)));

    CHECK_CUFFT_ERRORS(
        hipfftExecC2R(P->HyPlan, ((DEVICE_PTR_STRUCT.DATA)->yFFT_s),
            ((DEVICE_PTR_STRUCT.DATA)->Outy)));

    CHECK_CUFFT_ERRORS(
        hipfftExecC2R(P->HzPlan, ((DEVICE_PTR_STRUCT.DATA)->zFFT_s),
            ((DEVICE_PTR_STRUCT.DATA)->Outz)));
}
__host__ void DemagFieldInverseFFT(PLANS P)
{
    CHECK_CUFFT_ERRORS(
        hipfftExecZ2D(P->HxPlan, (hipfftDoubleComplex*)((DEVICE_PTR_STRUCT.DATA)->xFFT),
            (double*)((DEVICE_PTR_STRUCT.DATA)->Outx_d)));

    CHECK_CUFFT_ERRORS(
        hipfftExecZ2D(P->HyPlan, (hipfftDoubleComplex*)((DEVICE_PTR_STRUCT.DATA)->yFFT),
            (double*)((DEVICE_PTR_STRUCT.DATA)->Outy_d)));

    CHECK_CUFFT_ERRORS(
        hipfftExecZ2D(P->HzPlan, (hipfftDoubleComplex*)((DEVICE_PTR_STRUCT.DATA)->zFFT),
            (double*)((DEVICE_PTR_STRUCT.DATA)->Outz_d)));
}
__host__ void MagnetisationFFT(PLANS P)
{
    CHECK_CUFFT_ERRORS(
        hipfftExecD2Z(P->MxPlan, ((DEVICE_PTR_STRUCT.DATA)->Outx_d),
            (hipfftDoubleComplex*)((DEVICE_PTR_STRUCT.DATA)->xFFT)));

    CHECK_CUFFT_ERRORS(
        hipfftExecD2Z(P->MyPlan, ((DEVICE_PTR_STRUCT.DATA)->Outy_d),
            (hipfftDoubleComplex*)(DEVICE_PTR_STRUCT.DATA)->yFFT));

    CHECK_CUFFT_ERRORS(
        hipfftExecD2Z(P->MzPlan, ((DEVICE_PTR_STRUCT.DATA)->Outz_d),
            (hipfftDoubleComplex*)((DEVICE_PTR_STRUCT.DATA))->zFFT));
}
__host__ void MagnetisationFFT_SinglePrecision_R2C(PLANS P)
{
    
    CHECK_CUFFT_ERRORS(
        hipfftExecR2C(P->MxPlan, ((DEVICE_PTR_STRUCT.DATA)->Outx),
            ((DEVICE_PTR_STRUCT.DATA)->xFFT_s)));

    CHECK_CUFFT_ERRORS(
        hipfftExecR2C(P->MyPlan, ((DEVICE_PTR_STRUCT.DATA)->Outy),
            (DEVICE_PTR_STRUCT.DATA)->yFFT_s));

    CHECK_CUFFT_ERRORS(
        hipfftExecR2C(P->MzPlan, ((DEVICE_PTR_STRUCT.DATA)->Outz),
            ((DEVICE_PTR_STRUCT.DATA))->zFFT_s));

}
__host__ void DemagTensorFFT_Symmetries(MEMDATA DATA_h, MEMDATA DATA_d)
{
    hipfftHandle KxxPlan;
    int Nx = 2 * NUM_h;
    int Ny = 2 * NUMY_h;
    int Nz = 2 * NUMZ_h;
    if (IsPBCEnabled == false)
    {
        Nx = 2 * NUM_h;
        Ny = 2 * NUMY_h;
        Nz = 2 * NUMZ_h;
    }
    if (PBC_x_h != 0)
    {
        Nx = NUM_h;
        Ny = 2 * NUMY_h;
        Nz = 2 * NUMZ_h;
    }
    if (PBC_y_h != 0)
    {
        Nx = 2 * NUM_h;
        Ny = NUMY_h;
        Nz = 2 * NUMZ_h;
    }
    if (PBC_y_h != 0 && PBC_x_h != 0)
    {
        Nx = NUM_h;
        Ny = NUMY_h;
        Nz = 2 * NUMZ_h;
    }

    CHECK_CUFFT_ERRORS(
        hipfftPlan3d(&KxxPlan, Nx, Ny, Nz, HIPFFT_Z2Z));

    int SIZE = Nx * Ny * Nz * sizeof(fftw_complex);

    MEMDATA temp = (MEMDATA)malloc(sizeof(Dataptr));
    PointerCheck(temp != NULL);

    checkCudaErrors(hipMemcpy(temp, DATA_d, sizeof(Dataptr), hipMemcpyDeviceToHost));

    checkCudaErrors(hipMemcpy(temp->xFFT, DATA_h->kxx, SIZE, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(temp->yFFT, DATA_h->kyy, SIZE, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(temp->zFFT, DATA_h->kzz, SIZE, hipMemcpyHostToDevice));

    //Diagonals

    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());

    CHECK_CUFFT_ERRORS(
        hipfftExecZ2Z(KxxPlan, (hipfftDoubleComplex*)((DEVICE_PTR_STRUCT.DATA)->xFFT),
            (hipfftDoubleComplex*)((DEVICE_PTR_STRUCT.DATA)->xFFT), HIPFFT_FORWARD));
    CHECK_CUFFT_ERRORS(
        hipfftExecZ2Z(KxxPlan, (hipfftDoubleComplex*)((DEVICE_PTR_STRUCT.DATA)->yFFT),
            (hipfftDoubleComplex*)((DEVICE_PTR_STRUCT.DATA)->yFFT), HIPFFT_FORWARD));
    CHECK_CUFFT_ERRORS(
        hipfftExecZ2Z(KxxPlan, (hipfftDoubleComplex*)((DEVICE_PTR_STRUCT.DATA)->zFFT),
            (hipfftDoubleComplex*)((DEVICE_PTR_STRUCT.DATA)->zFFT), HIPFFT_FORWARD));

    DemagTensorStoreFirstOctant_Diagonals << <NumberofBlocksPadded, NumberofThreadsPadded >> > (DATA_d);

    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //Off-Diagonals

    checkCudaErrors(hipMemcpy(temp->xFFT, DATA_h->kxy, SIZE, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(temp->yFFT, DATA_h->kxz, SIZE, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(temp->zFFT, DATA_h->kyz, SIZE, hipMemcpyHostToDevice));

    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());

    CHECK_CUFFT_ERRORS(
        hipfftExecZ2Z(KxxPlan, (hipfftDoubleComplex*)((DEVICE_PTR_STRUCT.DATA)->xFFT),
            (hipfftDoubleComplex*)((DEVICE_PTR_STRUCT.DATA)->xFFT), HIPFFT_FORWARD));
    CHECK_CUFFT_ERRORS(
        hipfftExecZ2Z(KxxPlan, (hipfftDoubleComplex*)((DEVICE_PTR_STRUCT.DATA)->yFFT),
            (hipfftDoubleComplex*)((DEVICE_PTR_STRUCT.DATA)->yFFT), HIPFFT_FORWARD));
    CHECK_CUFFT_ERRORS(
        hipfftExecZ2Z(KxxPlan, (hipfftDoubleComplex*)((DEVICE_PTR_STRUCT.DATA)->zFFT),
            (hipfftDoubleComplex*)((DEVICE_PTR_STRUCT.DATA)->zFFT), HIPFFT_FORWARD));

    DemagTensorStoreFirstOctant_OffDiagonals << <NumberofBlocksPadded, NumberofThreadsPadded >> > (DATA_d);

    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());

    free(temp);
    hipfftDestroy(KxxPlan);

}